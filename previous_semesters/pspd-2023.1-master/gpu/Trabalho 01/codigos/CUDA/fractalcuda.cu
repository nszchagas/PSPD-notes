#include "hip/hip_runtime.h"
/*
    PSPD 2023 - 1
    Alunos: Antonio Aldisio        202028211
            Fernando Miranda Calil 190106565
            Lorrany Oliveira Souza 180113992

    Como compilar:
    $ nvcc fractalcuda.cu -o fractalcuda -lm

    Como rodar:
    $ cuda-memcheck ./fractalcuda 1000

*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#define OUTFILE "out_julia_normal_cu.bmp"

__device__ int compute_julia_pixel(int x, int y, int largura, int altura, float tint_bias, unsigned char *rgb) {
  // Check coordinates
  if ((x < 0) || (x >= largura) || (y < 0) || (y >= altura)) {
    printf("Invalid (%d,%d) pixel coordinates in a %d x %d image\n", x, y, largura, altura);
    return -1;
  }
  // "Zoom in" to a pleasing view of the Julia set
  float X_MIN = -1.6, X_MAX = 1.6, Y_MIN = -0.9, Y_MAX = +0.9;
  float float_y = (Y_MAX - Y_MIN) * (float)y / altura + Y_MIN ;
  float float_x = (X_MAX - X_MIN) * (float)x / largura  + X_MIN ;
  // Point that defines the Julia set
  float julia_real = -.79;
  float julia_img = .15;
  // Maximum number of iteration
  int max_iter = 300;
  // Compute the complex series convergence
  float real=float_y, img=float_x;
  int num_iter = max_iter;
  while (( img * img + real * real < 2 * 2 ) && ( num_iter > 0 )) {
    float xtemp = img * img - real * real + julia_real;
    real = 2 * img * real + julia_img;
    img = xtemp;
    num_iter--;
  }

  // Paint pixel based on how many iterations were used, using some funky colors
  float color_bias = (float) num_iter / max_iter;
  rgb[0] = (num_iter == 0 ? 200 : - 500.0 * pow(tint_bias, 1.2) *  pow(color_bias, 1.6));
  rgb[1] = (num_iter == 0 ? 100 : -255.0 *  pow(color_bias, 0.3));
  rgb[2] = (num_iter == 0 ? 100 : 255 - 255.0 * pow(tint_bias, 1.2) * pow(color_bias, 3.0));

  return 0;
}

__global__ void compute_julia_pixels(unsigned char *pixel_array, int largura, int altura) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int area = largura * altura;
  if (x < largura && y < altura) {
    int local_i = (y * largura + x) * 3;
    unsigned char rgb[3];
    compute_julia_pixel(x, y, largura, altura, 1.0, rgb);
    pixel_array[local_i] = rgb[0];
    pixel_array[local_i + 1] = rgb[1];
    pixel_array[local_i + 2] = rgb[2];
  }
}

int write_bmp_header(FILE *f, int largura, int altura) {
  unsigned int row_size_in_bytes = largura * 3 + ((largura * 3) % 4 == 0 ? 0 : (4 - (largura * 3) % 4));

  // Define all fields in the bmp header
    char id[3] = "BM"; // Alterado o tamanho para 3
    unsigned int filesize = 54 + (int)(row_size_in_bytes * altura * sizeof(char));
    short reserved[2] = {0, 0};
    unsigned int offset = 54;

    unsigned int size = 40;
    unsigned short planes = 1;
    unsigned short bits = 24;
    unsigned int compression = 0;
    unsigned int image_size = largura * altura * 3 * sizeof(char);
    int x_res = 0;
    int y_res = 0;
    unsigned int ncolors = 0;
    unsigned int importantcolors = 0;


  // Write the bytes to the file, keeping track of the
  // number of written "objects"
  size_t ret = 0;
  ret += fwrite(id, sizeof(char), 2, f);
  ret += fwrite(&filesize, sizeof(int), 1, f);
  ret += fwrite(reserved, sizeof(short), 2, f);
  ret += fwrite(&offset, sizeof(int), 1, f);
  ret += fwrite(&size, sizeof(int), 1, f);
  ret += fwrite(&largura, sizeof(int), 1, f);
  ret += fwrite(&altura, sizeof(int), 1, f);
  ret += fwrite(&planes, sizeof(short), 1, f);
  ret += fwrite(&bits, sizeof(short), 1, f);
  ret += fwrite(&compression, sizeof(int), 1, f);
  ret += fwrite(&image_size, sizeof(int), 1, f);
  ret += fwrite(&x_res, sizeof(int), 1, f);
  ret += fwrite(&y_res, sizeof(int), 1, f);
  ret += fwrite(&ncolors, sizeof(int), 1, f);
  ret += fwrite(&importantcolors, sizeof(int), 1, f);

  // Success means that we wrote 17 "objects" successfully
  return (ret != 17);
} /* fim write bmp-header */

int main(int argc, char *argv[]) {
  int n;
  int area = 0, largura = 0, altura = 0;
  FILE *output_file;
  unsigned char *pixel_array, *d_pixel_array;

  if ((argc <= 1) || (atoi(argv[1]) < 1)) {
    fprintf(stderr, "Entre 'N' como um inteiro positivo! \n");
    return -1;
  }
  n = atoi(argv[1]);
  altura = n; largura = 2 * n; area = altura * largura * 3;

  // Allocate host memory for the pixel array
  pixel_array = (unsigned char*)malloc(area * sizeof(unsigned char));

  // Allocate device memory for the pixel array
  hipMalloc((void**)&d_pixel_array, area * sizeof(unsigned char));

  dim3 threadsPerBlock(2, 2);
  dim3 numBlocks((largura + threadsPerBlock.x - 1) / threadsPerBlock.x, (altura + threadsPerBlock.y - 1) / threadsPerBlock.y);

  printf("Computando linhas de pixel %d até %d, para uma área total de %d\n", 0, n-1, area);

  // Copy the pixel array from host to device
  hipMemcpy(d_pixel_array, pixel_array, area * sizeof(unsigned char), hipMemcpyHostToDevice);

  // Start measuring the execution time
  clock_t start = clock();

  // Launch the kernel
  compute_julia_pixels<<<numBlocks, threadsPerBlock>>>(d_pixel_array, largura, altura);

  // Synchronize to make sure the kernel finishes execution
  hipDeviceSynchronize();

  // Stop measuring the execution time
  clock_t end = clock();

  // Calculate the elapsed time in seconds
  double elapsed_time = (double)(end - start) / CLOCKS_PER_SEC;
  printf("Tempo de execucao: %.4f segundos\n", elapsed_time);

  // Copy the pixel array from device to host
  hipMemcpy(pixel_array, d_pixel_array, area * sizeof(unsigned char), hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(d_pixel_array);

  // Write the BMP file
  output_file = fopen(OUTFILE, "w");
  write_bmp_header(output_file, largura, altura);
  fwrite(pixel_array, sizeof(unsigned char), area, output_file);
  fclose(output_file);

  // Free host memory
  free(pixel_array);

  return 0;
}