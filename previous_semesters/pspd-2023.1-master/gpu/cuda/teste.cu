
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>

#define N 100000
#define MAX_ERR 


__global__ void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}


int main(){
    float *a, *b, *out; 

    // Allocate memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    // Main function
    vector_add<<<1,1024>>>(out, a, b, N);

    // Verification
  hipDeviceSynchronize();
    printf("out[0] = %f\n", out[N]);
    printf("PASSED\n");
}
