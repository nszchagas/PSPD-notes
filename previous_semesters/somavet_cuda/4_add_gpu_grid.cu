
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline hipError_t checkCuda(hipError_t result) {
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

__global__ void add( int N, float *a, float *b, float *c ) {
    int tid = blockIdx.x*blockDim.x + threadIdx.x;    // this thread handles the data at its thread id

	printf("blockDim.x = %d, blockIdx.x=%d, threadIdx.x=%d\n", blockDim.x, blockIdx.x, threadIdx.x);

    if (tid < N)
        c[tid] = a[tid] + b[tid];
}

int main(int argc, char* argv[]) {
    if (argc != 4) {
      fprintf(stderr, "Syntax: %s <vector size N> <block size> <device id>\n", argv[0]);
      return EXIT_FAILURE;
    }

    int N = atoi(argv[1]);
    int BlockSize = atoi(argv[2]);
    int devId = atoi(argv[3]);

    checkCuda( hipSetDevice(devId) );

    float *a, *b, *c;
    // allocate the memory on the CPU
    a = (float *)malloc( N * sizeof (float));
    b = (float *)malloc( N * sizeof (float));
    c = (float *)malloc( N * sizeof (float));

    float *dev_a, *dev_b, *dev_c;
    // allocate the memory on the GPU
    checkCuda( hipMalloc( (void**)&dev_a, N * sizeof(float) ) );
    checkCuda( hipMalloc( (void**)&dev_b, N * sizeof(float) ) );
    checkCuda( hipMalloc( (void**)&dev_c, N * sizeof(float) ) );

    // fill the arrays 'a' and 'b' on the CPU
    for (int i=0; i<N; i++) {
        a[i] = -i;
        b[i] = i * i;
    }

    // copy the arrays 'a' and 'b' to the GPU
    checkCuda( hipMemcpy( dev_a, a, N * sizeof(float), hipMemcpyHostToDevice ) );
    checkCuda( hipMemcpy( dev_b, b, N * sizeof(float), hipMemcpyHostToDevice ) );

    int GridSize = (N+BlockSize-1)/BlockSize;
    add<<< GridSize, BlockSize >>>( N, dev_a, dev_b, dev_c );

    // copy the array 'c' back from the GPU to the CPU
    checkCuda( hipMemcpy( c, dev_c, N * sizeof(float), hipMemcpyDeviceToHost ) );  
    
    // display the results
//    for (int i=0; i<N; i++) {
//        printf( "%6.0f + %6.0f = %6.0f\n", a[i], b[i], c[i] );
//    }

    hipDeviceProp_t prop;
    checkCuda( hipGetDeviceProperties(&prop, devId));
    printf("Device: %s\n", prop.name);   

    // free the memory allocated on the CPU
    free(a); a=NULL;
    free(b); b=NULL;
    free(c); c=NULL;

    // free the memory allocated on the GPU
    checkCuda( hipFree( dev_a ) );
    checkCuda( hipFree( dev_b ) );
    checkCuda( hipFree( dev_c ) );

    return 0;
}
