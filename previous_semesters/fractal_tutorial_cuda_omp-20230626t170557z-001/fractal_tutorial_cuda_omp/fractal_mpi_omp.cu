#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <mpi.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <string.h>

#define OUTFILE "out_julia_normal_mpi_omp_cuda.bmp"


__device__ void compute_julia_pixel(int x, int y, int largura, int altura, float tint_bias, unsigned char *pixel_array, int index)
{
  // "Zoom in" to a pleasing view of the Julia set
  float X_MIN = -1.6, X_MAX = 1.6, Y_MIN = -0.9, Y_MAX = +0.9;
  float float_y = (Y_MAX - Y_MIN) * (float)y / altura + Y_MIN;
  float float_x = (X_MAX - X_MIN) * (float)x / largura + X_MIN;
  // Point that defines the Julia set
  float julia_real = -.79;
  float julia_img = .15;
  // Maximum number of iteration
  int max_iter = 300;
  // Compute the complex series convergence
  float real = float_y, img = float_x;
  int num_iter = max_iter;
  while ((img * img + real * real < 2 * 2) && (num_iter > 0))
  {
    float xtemp = img * img - real * real + julia_real;
    real = 2 * img * real + julia_img;
    img = xtemp;
    num_iter--;
  }

  // Paint pixel based on how many iterations were used, using some funky colors
  float color_bias = (float)num_iter / max_iter;
  pixel_array[index] = (num_iter == 0 ? 200 : (char)(-500.0 * pow(tint_bias, 1.2) * pow(color_bias, 1.6)));
  pixel_array[index + 1] = (num_iter == 0 ? 100 : (char)(-255.0 * pow(color_bias, 0.3)));
  pixel_array[index + 2] = (num_iter == 0 ? 100 : (char)(255 - 255.0 * pow(tint_bias, 1.2) * pow(color_bias, 3.0)));
} /*fim compute julia pixel */

void compute_julia_pixel2(int x, int y, int largura, int altura, float tint_bias, unsigned char *rgb)
{
  // "Zoom in" to a pleasing view of the Julia set
  float X_MIN = -1.6, X_MAX = 1.6, Y_MIN = -0.9, Y_MAX = +0.9;
  float float_y = (Y_MAX - Y_MIN) * (float)y / altura + Y_MIN;
  float float_x = (X_MAX - X_MIN) * (float)x / largura + X_MIN;
  // Point that defines the Julia set
  float julia_real = -.79;
  float julia_img = .15;
  // Maximum number of iteration
  int max_iter = 300;
  // Compute the complex series convergence
  float real = float_y, img = float_x;
  int num_iter = max_iter;
  while ((img * img + real * real < 2 * 2) && (num_iter > 0))
  {
    float xtemp = img * img - real * real + julia_real;
    real = 2 * img * real + julia_img;
    img = xtemp;
    num_iter--;
  }

  // Paint pixel based on how many iterations were used, using some funky colors
  float color_bias = (float)num_iter / max_iter;
  rgb[0] = (num_iter == 0 ? 200 : (char)(-500.0 * pow(tint_bias, 1.2) * pow(color_bias, 1.6)));
  rgb[1] = (num_iter == 0 ? 100 : (char)(-255.0 * pow(color_bias, 0.3)));
  rgb[2] = (num_iter == 0 ? 100 : (char)(255 - 255.0 * pow(tint_bias, 1.2) * pow(color_bias, 3.0)));
} /*fim compute julia pixel */

__global__ void compute(int altura_inicio, int largura, int altura_total, unsigned char *pixel_array)
{
  int i = blockIdx.x + altura_inicio;
  unsigned char rgb1, rgb2, rgb3;
  int index = blockIdx.x * largura * 3;

  if (blockIdx.x >= altura_total) {
    return;
  }

  for (int j = 0; j < largura * 3; j += 3)
  {
    compute_julia_pixel(j / 3, i, largura, altura_total, 1.0, pixel_array, index + j);
  }
}

int write_bmp_header(MPI_File file, int largura, int altura)
{

  unsigned int row_size_in_bytes = largura * 3 +
                                   ((largura * 3) % 4 == 0 ? 0 : (4 - (largura * 3) % 4));

  // Define all fields in the bmp header
  char id[3] = "BM";
  unsigned int filesize = 54 + (int)(row_size_in_bytes * altura * sizeof(char));
  short reserved[2] = {0, 0};
  unsigned int offset = 54;

  unsigned int size = 40;
  unsigned short planes = 1;
  unsigned short bits = 24;
  unsigned int compression = 0;
  unsigned int image_size = largura * altura * 3 * sizeof(char);
  int x_res = 0;
  int y_res = 0;
  unsigned int ncolors = 0;
  unsigned int importantcolors = 0;

  // Write the bytes to the file, keeping track of the
  // number of written "objects"
  size_t ret = 0;
  MPI_Status status;
  ret += MPI_File_write(file, id, 2, MPI_CHAR, &status);
  ret += MPI_File_write(file, &filesize, 1, MPI_INT, &status);
  ret += MPI_File_write(file, reserved, 2, MPI_SHORT, &status);
  ret += MPI_File_write(file, &offset, 1, MPI_INT, &status);
  ret += MPI_File_write(file, &size, 1, MPI_INT, &status);
  ret += MPI_File_write(file, &largura, 1, MPI_INT, &status);
  ret += MPI_File_write(file, &altura, 1, MPI_INT, &status);
  ret += MPI_File_write(file, &planes, 1, MPI_SHORT, &status);
  ret += MPI_File_write(file, &bits, 1, MPI_SHORT, &status);
  ret += MPI_File_write(file, &compression, 1, MPI_INT, &status);
  ret += MPI_File_write(file, &image_size, 1, MPI_INT, &status);
  ret += MPI_File_write(file, &x_res, 1, MPI_INT, &status);
  ret += MPI_File_write(file, &y_res, 1, MPI_INT, &status);
  ret += MPI_File_write(file, &ncolors, 1, MPI_INT, &status);
  ret += MPI_File_write(file, &importantcolors, 1, MPI_INT, &status);
 
  // Success means that we wrote 17 "objects" successfully
  return (ret != 17);
} /* fim write bmp-header */

int main(int argc, char *argv[])
{
  int n;
  char hostname[100];
  unsigned char *pixel_array, rgb[3], *pixel_array_dev;
  int area = 0, largura = 0, altura = 0;
  int altura_node, altura_inicio, altura_fim;
  MPI_File file;

  int rank, group_size;

  if ((argc <= 1) || (atoi(argv[1]) < 1))
  {
    fprintf(stderr, "Entre 'N' como um inteiro positivo! \n");
    return -1;
  }

  MPI_Init(&argc, &argv);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  MPI_Comm_size(MPI_COMM_WORLD, &group_size);
  printf("%d\n", MPI_File_open(MPI_COMM_WORLD, OUTFILE, MPI_MODE_WRONLY | MPI_MODE_CREATE, MPI_INFO_NULL, &file));

  n = atoi(argv[1]);

  altura_node = n / group_size;
  altura_inicio = rank * altura_node;
  altura_fim = (rank + 1) == group_size ? n : altura_inicio + altura_node;
  altura = altura_fim - altura_inicio;
  largura = 2 * n;
  area = altura * largura * 3;

  pixel_array = (unsigned char *)malloc(area * sizeof(unsigned char));

  int hostname_len = 0;
  MPI_Get_processor_name(hostname, &hostname_len);
  printf("%s: Computando linhas de pixel %d até %d, para uma área total de %d\n",hostname, 0, n - 1, area);

  if (strstr(hostname, "gpu")) {
    hipMalloc(&pixel_array_dev, area * sizeof(unsigned char));

    compute<<<altura, 1>>>(altura_inicio, largura, n, pixel_array_dev);

    hipMemcpy(pixel_array, pixel_array_dev, area * sizeof(unsigned char), hipMemcpyDeviceToHost);
  } else {
#pragma omp parallel for shared(pixel_array) private(rgb)
    for (int i = altura_inicio; i < altura_fim; i++)
    {
      int index = (i - altura_inicio) * largura * 3;
      for (int j = 0; j < largura * 3; j += 3)
      {
        compute_julia_pixel2(j / 3, i, largura, n, 1.0, rgb);

        pixel_array[index + j] = rgb[0];
        pixel_array[index + j + 1] = rgb[1];
        pixel_array[index + j + 2] = rgb[2];
      }
    }
  }

  // escreve o cabeçalho do arquivo
  MPI_Status status;
  MPI_Offset offset = 54;

  if (rank == 0)
  {
    write_bmp_header(file, largura, n);
    MPI_File_write_at(file, offset, pixel_array, area, MPI_CHAR, &status);
  }
  else
  {
    offset += (altura_inicio - 1) * largura * 3;
    MPI_File_write_at(file, offset, pixel_array, area, MPI_CHAR, &status);
  }

  free(pixel_array);
  if (strstr(hostname, "gpu")) {
    hipFree(pixel_array_dev);
  }

  MPI_File_close(&file);
  MPI_Finalize();
  return 0;
}
